#include "hip/hip_runtime.h"
#include<iostream>
#include<omp.h>
#include<chrono>
#include<stdio.h>
#include<math.h>
#include<algorithm>
#include<immintrin.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STB_FAILURE_USERMSG
#include "stb_image.h"
#include "stb_image_write.h"

#define GPUDIM threadIdx.x + blockIdx.x * blockDim.x

struct Image {
  uint8_t *r, *g, *b;
  int x, y, n;
};

struct Pixel {
  double r, g, b;
};

int max(int a, int b){
  return (a<b)?b:a;
}

int min(int a, int b){
  return (a>b)?b:a;
}

void cpyPix(Pixel *p, Image *img, int x, int y){
  p->r = (double)img->r[y*(img->x)+x];
  p->g = (double)img->g[y*(img->x)+x];
  p->b = (double)img->b[y*(img->x)+x];
}

__global__ void rotateImage(Image *base, Image *result, double theta){

  int y = result->y, x = result->x;
  float cX = base->x/2.0, cY = base->y/2.0;
  float cTheta = cos(theta);
  float sTheta = sin(theta);

if((GPUDIM) > x*y){
  for(int i=0; i<y; i++){
    float yt = cY-(GPUDIM/x);
    for(int j=0; j<x; j++){
      float xt = j-(GPUDIM%x);
      
      float xprime = xt*cTheta+yt*sTheta;
      float yprime = yt*cTheta-xt*sTheta;
      
      xprime += cX;
      yprime += cY;
      
      int q0x, q0y, q1x, q1y, q2x, q2y, q3x, q3y;

      q0x = (int)floor(xprime);
      q0y = (int)floor(yprime);
      q0x = max(0, q0x);
      q0x = min(q0x, base->x-1);
      q0y = max(0, q0y);
      q0y = min(q0y, base->y-1);

      q1x = (int)ceil(xprime);
      q1x = max(0, q1x);
      q1x = min(q1x, base->x-1);
      q1y = q0y;

      q2x = q0x;
      q2y = ceil(yprime);
      q2y = max(0, q2y);
      q2y = min(q2y, base->y-1);
      q3x = q1x;
      q3y = q2y;

      float f1, f2, f3, f4;
      Pixel P, R1, R2, q0, q1, q2, q3;

      cpyPix(&q0, base, q0x, q0y);
      cpyPix(&q1, base, q1x, q1y);
      cpyPix(&q2, base, q2x, q2y);
      cpyPix(&q3, base, q3x, q3y);

      if(q3x == q2x){
	f1=1;
	f2=0;
      }else{
	f1 = ((q3x - xprime)/(q3x - q2x));
	f2 = ((xprime - q2x)/(q3x - q2x));
      }

      R1.r = q2.r*f1+q3.r*f2;
      R1.g = q2.g*f1+q3.g*f2;
      R1.b = q2.b*f1+q3.b*f2;

      R2.r = q0.r*f1+q1.r*f2;
      R2.g = q0.g*f1+q1.g*f2;
      R2.b = q0.b*f1+q1.b*f2;
      
      if(q0y==q2y){
	f3 = 1;
	f4 = 0;
      }else{
	f3 = (((q2y-yprime)/(q2y-q0y)));
	f4 = (((yprime-q0y)/(q2y-q0y)));
      }

      P.r = R1.r*f3 + R2.r*f4;
      P.g = R1.g*f3 + R2.g*f4;
      P.b = R1.b*f3 + R2.b*f4;

      P.r = min(255.0, P.r);
      P.r = max(P.r, 0.0);
      P.g = min(255.0, P.g);
      P.g = max(P.g, 0.0);
      P.b = min(255.0, P.b);
      P.b = max(P.b, 0.0);
      
      
	result->r[((y-1)-(GPUDIM/x)*x+(GPUDIM%x)]=(uint8_t)P.r;
	result->g[((y-1)-(GPUDIM/x)*x+(GPUDIM%x)]=(uint8_t)P.g;
	result->b[((y-1)-(GPUDIM/x)*x+(GPUDIM%x)]=(uint8_t)P.b;
	
      
    }
  }
  }
}
}

int main(int argc, char* argv[]){
  long long int x = 0, y = 0;
  int numRot, block;
  double theta = 0.;
  x = atoll(argv[1]);
  y = atoll(argv[2]);
  theta = atof(argv[3]);
  numRot = atoi(argv[4])*100;
  block = atoi(argv[5]);
  std::chrono::steady_clock::time_point start, end, start1, end1;
  
  Image baseImage = {};
  
  uint8_t* data = (uint8_t *)malloc(x*y*3*sizeof(uint8_t));
  
  for(long long int i=0; i<y; i++){
    for(long long int j=0; j<x; j++){
      if(i*x*3+j*3 < x*y*3/2){
	data[i*x*3+j*3] = 0;
	data[i*x*3+j*3+1] = 0;
	data[i*x*3+j*3+2] = 255;
      }else{
	data[i*x*3+j*3] = 255;
	data[i*x*3+j*3+1] = 0;
	data[i*x*3+j*3+2] = 0;
      }
    }
  }

  stbi_write_png("rotateme.png", x, y, 3, data, x*3);
  
  baseImage.r = (uint8_t *)malloc(y*x*sizeof(uint8_t));
  baseImage.g = (uint8_t *)malloc(y*x*sizeof(uint8_t));
  baseImage.b = (uint8_t *)malloc(y*x*sizeof(uint8_t));
  Image centered= {};
  Image d_centered = {};
  Image result= {};
  Image d_result={};
  
  for(long long int i=0; i<y; i++){
    for(long long int j=0; j<x; j++){
      baseImage.r[i*x+j] = data[i*x*3+j*3];
      baseImage.g[i*x+j] = data[i*x*3+j*3+1];
      baseImage.b[i*x+j] = data[i*x*3+j*3+2];
    }
  }
  baseImage.x = x;
  baseImage.y = y;
  baseImage.n = 3;

  
  theta *= M_PI/180;
  int maxwah = ceil(sqrt(x*x + y*y));
  int rotSizex = maxwah;
  int rotSizey = maxwah;
  std::cout<<rotSizex<<"\t"<<rotSizey<<"\n";
 
  
  
  centered.x = rotSizex;
  centered.y = rotSizey;
  centered.n = 3;
  hipMalloc(d_centered.r, rotSizex*rotSizey*sizeof(uint8_t));
  hipMalloc(d_centered.g, rotSizex*rotSizey*sizeof(uint8_t));
  hipMalloc(d_centered.b, rotSizex*rotSizey*sizeof(uint8_t));
  hipMalloc(d_centered.x, sizeof(int));
  hipMalloc(d_centered.y, sizeof(int));
  hipMalloc(d_centered.n, sizeof(int));
  hipMemcpy(d_centered.x, centered.x, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_centered.y, centered.y, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_centered.n, centered.n, sizeof(int), hipMemcpyHostToDevice);
  result.r = (uint8_t *)malloc(rotSizex*rotSizey*sizeof(uint8_t));
  result.g = (uint8_t *)malloc(rotSizex*rotSizey*sizeof(uint8_t));
  result.b = (uint8_t *)malloc(rotSizex*rotSizey*sizeof(uint8_t));
  result.x = rotSizex;
  result.y = rotSizey;
  result.n = 3;
  hipMalloc(d_result.r, rotSizex*rotSizey*sizeof(uint8_t));
  hipMalloc(d_result.g, rotSizex*rotSizey*sizeof(uint8_t));
  hipMalloc(d_result.b, rotSizex*rotSizey*sizeof(uint8_t));
  hipMalloc(d_result.x, sizeof(int));
  hipMalloc(d_result.y, sizeof(int));
  hipMalloc(d_result.n, sizeof(int));
  int offsetx = (centered.x/2-baseImage.x/2);
  int offsety = (centered.y/2-baseImage.y/2);
  start = std::chrono::steady_clock::now();
  for(int i=0; i<numRot; i++){
   #pragma omp parallel for
    for(long long int i=0;i<y; i++){
    for(long long int j=0; j<x; j++){
      if(((i+offsety)*rotSizex)+(j+offsetx)<rotSizex*rotSizey){
	centered.r[((i+offsety)*rotSizex)+(j+offsetx)] = baseImage.r[i*x+j];
        centered.g[((i+offsety)*rotSizex)+(j+offsetx)] = baseImage.g[i*x+j];
	centered.b[((i+offsety)*rotSizex)+(j+offsetx)] = baseImage.b[i*x+j];
      }else{
	centered.r[i*rotSizex*j] = 0;
	centered.g[i*rotSizex*j] = 0;
	centered.b[i*rotSizex*j] = 0;
      }
    }
  }

  hipMemcpy(d_centered.r, centered.r, y*x*sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(d_centered.g, centered.g, y*x*sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(d_centered.b, centered.b, y*x*sizeof(uint8_t), hipMemcpyHostToDevice);

  rotateImage<<<(centered.x*centered.y+(block-1))/block, block>>>(&d_centered, &d_result, theta);

  hipMemcpy(result.r, d_result.r, y*x*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy(result.g, d_result.g, y*x*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy(result.b, d_result.b, y*x*sizeof(uint8_t), hipMemcpyDeviceToHost);
  free(centered.r);
  free(centered.g);
  free(centered.b);
  free(result.r);
  free(result.g);
  free(result.b);
  hipFree(d_centered.r);
  hipFree(d_centered.g);
  hipFree(d_centered.b);
  hipFree(d_result.r);
  hipFree(d_result.g);
  hipFree(d_result.b);
}
  end = std::chrono::steady_clock::now();
/*
uint8_t* outdata = (uint8_t *)malloc(rotSizex*rotSizey*3*sizeof(uint8_t));
  for(long long int i=0; i<rotSizey; i++){
    for(long long int j=0; j<rotSizex; j++){
      outdata[i*rotSizex*3+j*3] = result.r[i*result.x+j];
      outdata[i*rotSizex*3+j*3+1] = result.g[i*result.x+j];
      outdata[i*rotSizex*3+j*3+2] = result.b[i*result.x+j];
      //std::cout<<(int)outdata[i*rotSizex*3+j*3]<<" "<<(int)outdata[i*rotSizex*3+j*3+1]<<" "<<(int)outdata[i*rotSizex*3+j*3+2]<<"\n";
    }
  }
  
  stbi_write_png("rotated.png", rotSizex, rotSizey, 3, outdata, rotSizex*3);
  */
  std::chrono::duration<double> rotate_time = std::chrono::duration_cast<std::chrono::duration<double>>(end-start);
  
  double rotInMs = rotate_time.count();
  std::cout<<"Rotation:\t"<<rotInMs<<std::endl;
  std::cerr<<numRot<<"\t"<<rotInMs<<std::endl;
  free(data);
  free(outdata);
  
  free(baseImage.r);
  free(baseImage.g);
  free(baseImage.b);
  
}

