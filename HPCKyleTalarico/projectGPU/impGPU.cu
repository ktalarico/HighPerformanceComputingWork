#include "hip/hip_runtime.h"
#include<iostream>
#include<omp.h>
#include<chrono>
#include<stdio.h>
#include<math.h>



#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STB_FAILURE_USERMSG
#include "stb_image.h"
#include "stb_image_write.h"

struct Image {
  uint8_t *r, *g, *b;
  int x, y, n;
};

struct Pixel {
  double r, g, b;
};

__device__ int maxNew(int a, int b){
  return (a<b)?b:a;
}

__device__ int minNew(int a, int b){
  return (a>b)?b:a;
}

__device__ void cpyPix(double p[3], Image img, int x, int y){
  p[0] = (double)img.r[y*(img.x)+x];
  p[1] = (double)img.g[y*(img.x)+x];
  p[2] = (double)img.b[y*(img.x)+x];
}

__global__ void rotateImage(Image base, Image result, double theta){
  int y = base.y, x = base.x;
  int dim = blockIdx.x * blockDim.x + threadIdx.x;
  float cX = x/2.0, cY = y/2.0;
  float cTheta = cos(theta);
  float sTheta = sin(theta);
  //printf("dim: %d\nblock: %d\n", dim, blockIdx.x);
  if(dim<x*y){
    float yt = cY-dim/x;
 
      float xt = dim%x-cX;
      //std::cout<<"xt: "<<xt<<"\tyt: "<<yt<<"\n";
      float xprime = xt*cTheta+yt*sTheta;
      float yprime = yt*cTheta-xt*sTheta;
      //std::cout<<"centx: "<<xprime<<"\tcenty: "<<yprime<<"\n";
      xprime += cX;
      yprime += cY;
      //std::cout<<"disx: "<<xprime<<"\tdisy: "<<yprime<<"\n";
      int q0x, q0y, q1x, q1y, q2x, q2y, q3x, q3y;
      
      q0x = (int)floor(xprime);
      q0y = (int)floor(yprime);
      q0x = maxNew(0, q0x);
      q0x = minNew(q0x, x-1);
      q0y = maxNew(0, q0y);
      q0y = minNew(q0y, y-1);

      q1x = (int)ceil(xprime);
      q1x = maxNew(0, q1x);
      q1x = minNew(q1x, x-1);
      q1y = q0y;

      q2x = q0x;
      q2y = ceil(yprime);
      q2y = maxNew(0, q2y);
      q2y = minNew(q2y, y-1);
      q3x = q1x;
      q3y = q2y;
      //std::cout<<"i: "<<i<<"\tj: "<<j<<"\n";
      //std::cout<<"xp: "<<xprime<<"\typ: "<<yprime<<"\n";
      //std::cout<<"00x: "<<q0x<<"\t"<<"01x: "<<q1x<<"\t"<<"10x: "<<q2x<<"\t"<<"11x: "<<q3x<<"\n"<<"00y: "<<q0y<<"\t"<<"01y: "<<q1y<<"\t"<<"10y: "<<q2y<<"\t"<<"11y: "<<q3y<<"\n";
      float f1, f2, f3, f4;
      double P[3], R1[3], R2[3], q0[3], q1[3], q2[3], q3[3];      

      cpyPix(q0, base, q0x, q0y);
      cpyPix(q1, base, q1x, q1y);
      cpyPix(q2, base, q2x, q2y);
      cpyPix(q3, base, q3x, q3y);
      //std::cout<<"q0r: "<<q0.r<<"\tq0g: "<<q0.g<<"\tq0b: "<<q0.b<<"\n";
      //std::cout<<"q1r: "<<q1.r<<"\tq1g: "<<q1.g<<"\tq1b: "<<q1.b<<"\n";
      //std::cout<<"q2r: "<<q2.r<<"\tq2g: "<<q2.g<<"\tq2b: "<<q2.b<<"\n";
      //std::cout<<"q3r: "<<q3.r<<"\tq3g: "<<q3.g<<"\tq3b: "<<q3.b<<"\n";
      if(q3x == q2x){
	f1=1;
	f2=0;
      }else{
	f1 = ((q3x - xprime)/(q3x - q2x));
	f2 = ((xprime - q2x)/(q3x - q2x));
      }

      R1[0] = q2[0]*f1+q3[0]*f2;
      R1[1] = q2[1]*f1+q3[1]*f2;
      R1[2] = q2[2]*f1+q3[2]*f2;

      R2[0] = q0[0]*f1+q1[0]*f2;
      R2[1] = q0[1]*f1+q1[1]*f2;
      R2[2] = q0[2]*f1+q1[2]*f2;
      //std::cout<<R1.r<<"\t"<<R1.g<<"\t"<<R1.b<<"\n";
      if(q0y==q2y){
	f3 = 1;
	f4 = 0;
      }else{
	f3 = (((q2y-yprime)/(q2y-q0y)));
	f4 = (((yprime-q0y)/(q2y-q0y)));
      }

      P[0] = R1[0]*f3 + R2[0]*f4;
      P[1] = R1[1]*f3 + R2[1]*f4;
      P[2] = R1[2]*f3 + R2[2]*f4;

      P[0] = minNew(255.0, P[0]);
      P[0] = maxNew(P[0], 0.0);
      P[1] = minNew(255.0, P[1]);
      P[1] = maxNew(P[1], 0.0);
      P[2] = minNew(255.0, P[2]);
      P[2] = maxNew(P[2], 0.0);
      //std::cout<<P.r<<"\t"<<P.g<<"\t"<<P.b<<"\n";
      //printf("y: %d\tx: %d\tdim: %d\n", dim/x, dim%x, dim);
	result.r[((y-1)-(dim/x))*x+(dim%x)]=(uint8_t)P[0];
	result.g[((y-1)-(dim/x))*x+(dim%x)]=(uint8_t)P[1];
	result.b[((y-1)-(dim/x))*x+(dim%x)]=(uint8_t)P[2];
	//std::cout<<(int)result->r[((y-1)-i)*x+(j)]<<"\t"<<(int)result->g[((y-1)-i)*x+(j)]<<"\t"<<(int)result->b[((y-1)-i)*x+(j)]<<"\n";
    
  
  }
}

int main(int argc, char* argv[]){
  
  long long int x = 0, y = 0;
  int numRot, blockSize=32, streamNum = 1;
  double theta = 0.;
  double allTheta[100];
  x = atoll(argv[1]);
  y = atoll(argv[2]);
  theta = atof(argv[3]);
  numRot = atoi(argv[4]);
  blockSize = atoi(argv[5]);
  streamNum = atoi(argv[6]);
  
  std::chrono::steady_clock::time_point start, end;
  
  Image baseImage = {};
  
  uint8_t* data = (uint8_t *)malloc(x*y*3*sizeof(uint8_t));
  
  for(int i=0; i<100; i++){
    allTheta[i] = theta;
  }

  for(long long int i=0; i<y; i++){
    for(long long int j=0; j<x; j++){
      if(i*x*3+j*3 < x*y*3/2){
	data[i*x*3+j*3] = 0;
	data[i*x*3+j*3+1] = 0;
	data[i*x*3+j*3+2] = 255;
      }else{
	data[i*x*3+j*3] = 255;
	data[i*x*3+j*3+1] = 0;
	data[i*x*3+j*3+2] = 0;
      }
    }
  }
  stbi_write_png("rotateme.png", x, y, 3, data, x*3);
  
  baseImage.r = (uint8_t *)malloc(y*x*sizeof(uint8_t));
  baseImage.g = (uint8_t *)malloc(y*x*sizeof(uint8_t));
  baseImage.b = (uint8_t *)malloc(y*x*sizeof(uint8_t));
  for(long long int i=0; i<y; i++){
    for(long long int j=0; j<x; j++){
      baseImage.r[i*x+j] = data[i*x*3+j*3];
      baseImage.g[i*x+j] = data[i*x*3+j*3+1];
      baseImage.b[i*x+j] = data[i*x*3+j*3+2];
    }
  }
  baseImage.x = x;
  baseImage.y = y;
  baseImage.n = 3;
  
  theta *= M_PI/180;
  int maxwah = ceil(sqrt(x*x + y*y));
  int rotSizex = maxwah;
  int rotSizey = maxwah;
  
  hipStream_t streams[streamNum];
  for(int i=0; i<streamNum; i++){
    hipStreamCreate(&streams[i]);
  }
  std::cout<<"streams"<<std::endl;
  Image centered= {};
  Image d_centered = {};
  std::cout<<"empty structs"<<std::endl;
  hipHostMalloc((void **)&(centered.r), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  hipHostMalloc((void **)&(centered.g), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  hipHostMalloc((void **)&(centered.b), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  centered.x = rotSizex;
  centered.y = rotSizey;
  centered.n = 3;
  
  hipMalloc((void **)&(d_centered.r), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif 
 hipMalloc((void **)&(d_centered.g), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  hipMalloc((void **)&(d_centered.b), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  d_centered.x = rotSizex;
  d_centered.y = rotSizey;
  d_centered.n = 3;
  Image result= {};
  Image d_result = {};
  hipHostMalloc((void **)&(result.r), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  hipHostMalloc((void **)&(result.g), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  hipHostMalloc((void **)&(result.b), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
  result.x = rotSizex;
  result.y = rotSizey;
  result.n = 3;
  
  hipMalloc((void **)&(d_result.r), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  hipMalloc((void **)&(d_result.g), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  hipMalloc((void **)&(d_result.b), rotSizex*rotSizey*sizeof(uint8_t));
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  d_result.x = rotSizex;
  d_result.y = rotSizey;
  d_result.n = 3;

  memset(centered.r, 0, rotSizex*rotSizey*sizeof(uint8_t));
  memset(centered.g, 0, rotSizex*rotSizey*sizeof(uint8_t));
  memset(centered.b, 0, rotSizex*rotSizey*sizeof(uint8_t));
  
  int offsetx = (centered.x/2-baseImage.x/2);
  int offsety = (centered.y/2-baseImage.y/2);
  
  start = std::chrono::steady_clock::now();
  for(int k=0; k<numRot; k++){
    #pragma omp parallel for collapse(2)
    for(long long int i=0;i<y; i++){
      for(long long int j=0; j<x; j++){
	centered.r[((i+offsety)*rotSizex)+(j+offsetx)] = baseImage.r[i*x+j];
        centered.g[((i+offsety)*rotSizex)+(j+offsetx)] = baseImage.g[i*x+j];
	centered.b[((i+offsety)*rotSizex)+(j+offsetx)] = baseImage.b[i*x+j];
      }
    }
   
  hipMemcpyAsync((void *)d_centered.r, (void *)centered.r, rotSizex*rotSizey*sizeof(uint8_t), hipMemcpyHostToDevice, streams[k%streamNum]);
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  hipMemcpyAsync((void *)d_centered.g, (void *)centered.g, rotSizex*rotSizey*sizeof(uint8_t), hipMemcpyHostToDevice, streams[k%streamNum]);
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  hipMemcpyAsync((void *)d_centered.b, (void *)centered.b, rotSizex*rotSizey*sizeof(uint8_t), hipMemcpyHostToDevice, streams[k%streamNum]);
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  for(int l=0; l<100; l++){
    rotateImage<<<((rotSizex*rotSizey)+(blockSize-1))/blockSize, blockSize, 0, streams[k%streamNum]>>>(d_centered, d_result, allTheta[l]);
  }
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  hipMemcpyAsync((void *)result.r, (void *)d_result.r, rotSizex*rotSizey*sizeof(uint8_t), hipMemcpyDeviceToHost, streams[k%streamNum]);
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  hipMemcpyAsync((void *)result.g, (void *)d_result.g, rotSizex*rotSizey*sizeof(uint8_t), hipMemcpyDeviceToHost, streams[k%streamNum]);
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
  hipMemcpyAsync((void *)result.b, (void *)d_result.b, rotSizex*rotSizey*sizeof(uint8_t), hipMemcpyDeviceToHost, streams[k%streamNum]);
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif  
}
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
#ifdef DEBUG 
std::cout<<__LINE__<<" Error: "<<hipGetErrorString(hipGetLastError())<<std::endl;
#endif
 uint8_t *outdata;
 outdata = (uint8_t *)malloc(rotSizex*rotSizey*3*sizeof(uint8_t));
  for(long long int i=0; i<rotSizey; i++){
    for(long long int j=0; j<rotSizex; j++){
      outdata[i*rotSizex*3+j*3] = result.r[i*result.x+j];
      outdata[i*rotSizex*3+j*3+1] = result.g[i*result.x+j];
      outdata[i*rotSizex*3+j*3+2] = result.b[i*result.x+j];
      //std::cout<<(int)outdata[i*rotSizex*3+j*3]<<" "<<(int)outdata[i*rotSizex*3+j*3+1]<<" "<<(int)outdata[i*rotSizex*3+j*3+2]<<"\n";
    }
  }
  //std::cout<<rotSizex<<"\t"<<rotSizey<<"\t"<<outdata[1]<<std::endl;
  stbi_write_png("rotated.png", rotSizex, rotSizey, 3, outdata, rotSizex*3);
  
  std::chrono::duration<double> rotate_time = std::chrono::duration_cast<std::chrono::duration<double>>(end-start);
  
  double rotInMs = rotate_time.count();
  std::cout<<"Rotation:\t"<<rotInMs<<std::endl;
  std::cerr<<numRot*100<<"\t"<<rotInMs<<std::endl;
  
  hipFree(d_centered.r);
  hipFree(d_centered.g);
  hipFree(d_centered.b);
  hipFree(d_result.r);
  hipFree(d_result.g);
  hipFree(d_result.b);
  free(data);
  free(outdata);
  hipHostFree(centered.r);
  hipHostFree(centered.g);
  hipHostFree(centered.b);
  hipHostFree(result.r);
  hipHostFree(result.g);
  hipHostFree(result.b);
  free(baseImage.r);
  free(baseImage.g);
  free(baseImage.b);
  
}

