
#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>

__global__ void expansion(float *a, float *poly, long long int n, long long int degree){
	
	if(threadIdx.x+blockIdx.x*blockDim.x < n){
	float temp = a[threadIdx.x+blockIdx.x*blockDim.x];
	float xpow = 1;
	for (int i=0; i<degree; i++){
      	    a[threadIdx.x+blockIdx.x*blockDim.x] += xpow*poly[i];
      	    xpow *= temp;
      	}
	}	
}

int main(int argc, char* argv[]){
    using namespace std::chrono;
    long long int size = atoi(argv[1]);
    long long int degree = atoi(argv[2]);
    int block = atoi(argv[3]);
    long long int bytepoly, byten;
    bytepoly = (degree+1)*sizeof(float);
    byten = size*sizeof(float);
    float *a, *d_a, *poly, *d_poly;
    long long int *d_degree;
    a = (float *)malloc(byten);
    poly = (float *)malloc(bytepoly);
    steady_clock::time_point start, end;
    hipMalloc((void **)&d_a, byten);
    hipMalloc((void **)&d_poly, bytepoly);
    hipMalloc((void **)&d_degree, sizeof(int));
    for(long long int i=0; i<size; i++){
       a[i] = 1.;
    }
    for(long long int i=0; i<degree+1; i++){
       poly[i] = 2.;
    }
    start = steady_clock::now();
    hipMemcpy(d_a, a, byten, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    end = steady_clock::now();
    hipMemcpy(d_poly, poly, bytepoly, hipMemcpyHostToDevice);
    expansion<<<((size+(block-1))/block),block>>>(d_a, d_poly, size, degree);
    printf("%s\n",hipGetErrorString(hipGetLastError()));
    hipMemcpy(a, d_a, byten, hipMemcpyDeviceToHost);
    duration<double> elapsed_time = duration_cast<duration<double>>(end-start);
    printf("%s\n",hipGetErrorString(hipGetLastError()));
    std::cout<<size<<"\t"<<degree<<"\t"<<size/elapsed_time.count()<<"\n";
}