
#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>

__global__ void expansion(float *a, float *poly, long long int n, long long int degree, int i, int streamSize, int block){
	
	if((i*streamSize*block)+threadIdx.x+blockIdx.x*blockDim.x < n){
	float temp = a[threadIdx.x+blockIdx.x*blockDim.x];
	float xpow = 1;
	for (long long int i=0; i<degree; i++){
      	    a[threadIdx.x+blockIdx.x*blockDim.x] += xpow*poly[i];
      	    xpow *= temp;
      	}
	}
}

int main(int argc, char* argv[]){
    using namespace std::chrono;
    long long int size = atoll(argv[1]);
    long long int degree = atoll(argv[2]);
    int block = atoi(argv[3]);
    int streamCnt = atoi(argv[4]);
    int streamSize = atoi(argv[5]);
    long long int byten, bytepoly;    
    
    float *a, *d_a, *poly, *d_poly;
    long long int numBlocks = (size+(block-1))/block;
    long long int streamBlk = (numBlocks+(streamSize-1))/streamSize;
    byten = streamSize*block*sizeof(float);
    bytepoly = (degree+1)*sizeof(float);
    
    hipStream_t streams[streamCnt];
    for(int i=0; i<streamCnt; i++){
    	    hipStreamCreate(&streams[i]);
    }
    hipHostMalloc((void **)&a, size*sizeof(float), hipHostMallocDefault);    
    hipHostMalloc((void **)&poly, bytepoly, hipHostMallocDefault);
    //std::cout<<__LINE__<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
    steady_clock::time_point start, end;
    
    hipMalloc((void **)&d_a, streamCnt*byten);
    hipMalloc((void **)&d_poly, bytepoly);
    //std::cout<<__LINE__<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
    for(long long int i=0; i<size; i++){
       a[i] = 1.;
    }
    for(long long int i=0; i<degree+1; i++){
       poly[i] = 1.;
    }
    start = steady_clock::now();
    hipMemcpyAsync(d_poly, poly, bytepoly, hipMemcpyHostToDevice);
    //std::cout<<__LINE__<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
    for(long long int i=0; i<streamBlk; i++){
    	if(i*streamSize*block+streamSize*block < size){    
	        hipMemcpyAsync(&d_a[i%streamCnt*streamSize*block], &a[i*streamSize*block], byten, hipMemcpyHostToDevice, streams[i%streamCnt]);
		//std::cout<<i<<"\t"<<i*streamSize*block<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
		expansion<<<streamSize, block, 0, streams[i%streamCnt]>>>(&d_a[i%streamCnt*streamSize*block], d_poly, size, degree, i, streamSize, block);
		//std::cout<<i<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
		hipMemcpyAsync(&a[i*streamSize*block], &d_a[i%streamCnt*streamSize*block], byten, hipMemcpyDeviceToHost, streams[i%streamCnt]);
		//std::cout<<i<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
	}else{
		hipMemcpyAsync(&d_a[i%streamCnt*streamSize*block], &a[i*streamSize*block], (size-i*streamSize*block)*sizeof(float), hipMemcpyHostToDevice, streams[i%streamCnt]);
		//std::cout<<i<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
		expansion<<<streamSize, block, 0, streams[i%streamCnt]>>>(&d_a[i%streamCnt*streamSize*block], d_poly, size, degree, i, streamSize, block);
		//std::cout<<i<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
		hipMemcpyAsync(&a[i*streamSize*block], &d_a[i%streamCnt*streamSize*block], (size-i*streamSize*block)*sizeof(float), hipMemcpyDeviceToHost, streams[i%streamCnt]);
		//std::cout<<i<<"\t"<<cudaGetErrorString(cudaGetLastError())<<std::endl;
    	}
    }
    hipDeviceSynchronize();
    std::cout<<__LINE__<<"\t"<<hipGetErrorString(hipGetLastError())<<std::endl;
    end = steady_clock::now();
    
    duration<double> elapsed_time = duration_cast<duration<double>>(end-start);
    std::cout<<size<<"\t"<<degree<<"\t"<<(size*degree*sizeof(float))/elapsed_time.count()<<std::endl;
    for(int i=0; i<streamCnt; i++){
    hipStreamDestroy(streams[i]);
    }
    hipFree(d_a);
    hipFree(d_poly);
}